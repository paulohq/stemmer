
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stddef.h>
#include <host_defines.h>




/**
 * Determina a quantidade de bytes que compõem uma string codificada em ASCII.
 *
 * @param s String codificada em UTF-8.
 * @return
 */
__device__
        size_t d_strlen (char* s)
{
    char* nav = s;

    while (*nav)
    {
        ++nav;
    }

    return (nav - s);
}



/**
 * Copia 'n' caracteres da string 'from' para o buffer apontado por 'to'.
 *
 * @param to Ponteiro para a região da memória que receberá a string.
 * @param from Ponteiro para a string que será copiada.
 * @param n Número máximo de caracteres que deve ser copiado.
 * @return
 */
__device__
char* d_strncpy (char* to, const char* from, size_t n)
{
    char* s = to;

    while ((n > 0) && (*from != '\0'))
    {
        *s++ = *from++;
        --n;
    }

    while (n > 0)
    {
        *s++ = '\0';
        --n;
    }

    return to;
}



/**
 * Determina se um caractere qualquer é uma letra ou não.
 *
 * Obs.: implementação para a GPU da função 'isalpha' disponível na STDLIB, mas indisponível no CUDART.
 *
 * @param c Caractere que será analisado.
 * @return
 */
__device__
bool d_isalpha (char c)
{
    return ((c >= 'a' && c <= 'z') || (c >= 'A' && c <= 'Z'));
}



/**
 * Determina se um caractere qualquer é um dígito ou não.
 *
 * Obs.: implementação para a GPU da função 'isdigit' disponível na STDLIB, mas indisponível no CUDART.
 *
 * @param c Caractere que será analisado.
 * @return
 */
__device__
bool d_isdigit (char c)
{
    return (c >= '0' && c <= '9');
}



/**
 * Converte um caractere para seu equivalente em minúsculo.
 *
 * @param c Caractere que será analisado.
 * @return Letra equivalente em minúsculo caso seja uma letra (A-Z), ou o próprio char caso não seja uma letra.
 */
__device__
char d_tolower (char c)
{
    if ((c >= 'A') && (c <= 'Z'))
    {
        return (char) (c + 32);
    }
    else
    {
        return c;
    }
}


/**
 * Procura uma substring dentro de uma string qualquer.
 *
 * @param string String que será usada para a busca.
 * @param substring Substring que deve ser localizada.
 * @return
 */
__device__
char* d_strstr (const char* string, const char* substring)
{
    char* nav_str, * nav_sub;
    //char c;

    nav_sub = (char*) substring;

    if (*nav_sub == '\0')
    {
        return (char*) string;
    }

    while (*string != '\0')
    {
        //
        // Procura pela coincidência do primeiro caractere.
        //
        if (*string++ != *nav_sub)
        {
            continue;
        }

        nav_str = (char*) string;

        while (true)
        {
            if (*nav_sub == '\0')
            {
                //
                // Localizamos a substring dentro da string. Retorna o valor atual do ponteiro.
                //
                return (char *) string;
            }
            else if (*nav_str++ != *nav_sub++)
            {
                break;
            }
        }

        //
        // Restaura o ponteiro para continuarmos procurando...
        //
        nav_sub = (char*) substring;
    }


    //return nullptr;
    return 0;
}

/**
 * Compara 'n' caracteres das duas strings passadas como parâmetro.
 *
 * @param string Ponteiro para a região da memória da primeira string.
 * @param string1 Ponteiro para a segunda string que será comparada.
 * @param n Número máximo de caracteres que deve ser comparado.
 * @return
 */
__device__
bool d_strncmp (char* string, const char* string1, size_t n)
{

    while ((n > 0) && (*string != '\0'))
    {
        if (string[n] != string1[n])
            return false;
        --n;
    }

    return true;
}
/*int strncmp(const char* s1, const char* s2, size_t n)
{
    while(n--)
        if(*s1++!=*s2++)
            return *(unsigned char*)(s1 - 1) - *(unsigned char*)(s2 - 1);
    return 0;
}*/

/**
 * Concatena uma string em outra de acordo com o tamanho
 *
 * @param to Ponteiro para a região da memória que receberá a string.
 * @param from Ponteiro para a string que será copiada.
 * @param n Número máximo de caracteres que deve ser copiado.
 * @return
 */
__device__
char *d_strncat(char *to, const char *from, size_t n)
{
    char *ret = to;
    while (*to)
        to++;
    while (n--)
        if (!(*to++ = *from++))
            return ret;
    *to = 0;
    return ret;
}
/*
char *d_strncat(char *dest, const char *src, size_t n)
{
    char *ret = dest;
    while (*dest)
        dest++;
    while (n--)
        if (!(*dest++ = *src++))
            return ret;
    *dest = 0;
    return ret;
}*/
